#include "hip/hip_runtime.h"
#include <iostream>
#include <stdint.h>
#include "run_udf.cuh"
#include <hip/hip_runtime.h>

__global__ void profile_udf_kernel(int* input, int* output, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N) return;

    output[tid] = run_udf(input[tid]);
}

int main() {
    const int N = 64;
    int h_input[N];
    int h_output[N];

    for (int i = 0; i < N; ++i)
        h_input[i] = i;

    int* d_input;
    int* d_output;
    hipMalloc(&d_input, sizeof(int) * N);
    hipMalloc(&d_output, sizeof(int) * N);

    hipMemcpy(d_input, h_input, sizeof(int) * N, hipMemcpyHostToDevice);

    profile_udf_kernel<<<1, N>>>(d_input, d_output, N);
    hipMemcpy(h_output, d_output, sizeof(int) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i)
        std::cout << "Thread " << i << " output: " << h_output[i] << std::endl;

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
