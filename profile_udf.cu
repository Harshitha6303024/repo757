#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>      // For std::setw and std::setprecision
#include <cstdlib>
#include <ctime>
#include <stdint.h>
#include "run_udf.cuh"

#define WARP_SIZE 32

__global__ void profile_udf_kernel(
    int* input, int* output, int* active_cycles, uint64_t* total_cycles, int N) {
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N) return;

    uint64_t start = clock64();

    int sum, count, active;
    run_udf(input[tid], &sum, &count, &active);

    uint64_t end = clock64();

    output[tid] = sum;
    active_cycles[tid] = active;
    total_cycles[tid] = end - start;
}

int main() {
    const int N = 32; // one warp
    int h_input[N], h_output[N], h_active_cycles[N];
    uint64_t h_total_cycles[N];

    srand(time(NULL));
    for (int i = 0; i < N; ++i)
        h_input[i] = rand() % 1000 + 100;  // Induce divergence

    int *d_input, *d_output, *d_active_cycles;
    uint64_t* d_total_cycles;

    hipMalloc(&d_input, sizeof(int) * N);
    hipMalloc(&d_output, sizeof(int) * N);
    hipMalloc(&d_active_cycles, sizeof(int) * N);
    hipMalloc(&d_total_cycles, sizeof(uint64_t) * N);

    hipMemcpy(d_input, h_input, sizeof(int) * N, hipMemcpyHostToDevice);

    profile_udf_kernel<<<1, N>>>(d_input, d_output, d_active_cycles, d_total_cycles, N);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(h_active_cycles, d_active_cycles, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(h_total_cycles, d_total_cycles, sizeof(uint64_t) * N, hipMemcpyDeviceToHost);

    // Find max active_cycles to normalize work scores
    int max_active = 0;
    for (int i = 0; i < N; ++i) {
        if (h_active_cycles[i] > max_active)
            max_active = h_active_cycles[i];
    }

    std::cout << "\n=== Per-Thread Execution Report (Integer Work Score) ===\n";
    for (int i = 0; i < N; ++i) {
        int work_value = h_active_cycles[i]; // Raw active cycles
        int work_score = (int)(((double)work_value / max_active) * 100.0);  // Score from 0–100
        double utilization = (double)h_active_cycles[i] / h_total_cycles[i] * 100.0;

        std::cout << "Thread " << std::setw(2) << i
                  << " | Input: " << std::setw(4) << h_input[i]
                  << " | Output: " << std::setw(9) << h_output[i]
                  << " | Work Value: " << std::setw(4) << work_value
                  << " | Work Score: " << std::setw(3) << work_score << "/100"
                  << " | Utilization: " << std::fixed << std::setprecision(2) << utilization << " %\n";
    }

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_active_cycles);
    hipFree(d_total_cycles);
    return 0;
}
