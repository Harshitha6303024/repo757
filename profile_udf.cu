#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <stdint.h>
#include "run_udf.cuh"

#define WARP_SIZE 32

__global__ void profile_udf_kernel(
    int* input, int* output, int* active_cycles, uint64_t* total_cycles, int N) {
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N) return;

    uint64_t start = clock64();

    int sum, count, active;
    run_udf(input[tid], &sum, &count, &active);

    uint64_t end = clock64();

    output[tid] = sum;
    active_cycles[tid] = active;
    total_cycles[tid] = end - start;
}

int main() {
    const int N = 64;
    int h_input[N], h_output[N], h_active_cycles[N];
    uint64_t h_total_cycles[N];

    srand(time(NULL));
    for (int i = 0; i < N; ++i)
        h_input[i] = rand() % 1000 + 100;  // Induce divergence

    int *d_input, *d_output, *d_active_cycles;
    uint64_t* d_total_cycles;

    hipMalloc(&d_input, sizeof(int) * N);
    hipMalloc(&d_output, sizeof(int) * N);
    hipMalloc(&d_active_cycles, sizeof(int) * N);
    hipMalloc(&d_total_cycles, sizeof(uint64_t) * N);

    hipMemcpy(d_input, h_input, sizeof(int) * N, hipMemcpyHostToDevice);

    profile_udf_kernel<<<1, N>>>(d_input, d_output, d_active_cycles, d_total_cycles, N);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(h_active_cycles, d_active_cycles, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(h_total_cycles, d_total_cycles, sizeof(uint64_t) * N, hipMemcpyDeviceToHost);

    // Find max active_cycles to normalize work values
    int max_active = 0;
    for (int i = 0; i < N; ++i) {
        if (h_active_cycles[i] > max_active)
            max_active = h_active_cycles[i];
    }

    std::cout << "\n=== Per-Thread Execution Report (with Work Value) ===\n";
    for (int i = 0; i < N; ++i) {
        double utilization = (double)h_active_cycles[i] / h_total_cycles[i] * 100.0;
        double normalized_work = (double)h_active_cycles[i] / max_active;

        std::cout << "Thread " << i
                  << " | Input: " << h_input[i]
                  << " | Output: " << h_output[i]
                  << " | Work Value (Active Cycles): " << h_active_cycles[i]
                  << " | Normalized Work: " << normalized_work
                  << " | Utilization: " << utilization << " %\n";
    }

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_active_cycles);
    hipFree(d_total_cycles);
    return 0;
}
