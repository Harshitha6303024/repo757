#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <stdint.h>
#include "run_udf_divergent.cuh"

#define WARP_SIZE 32

__global__ void profile_udf_kernel(int* input, int* output, uint64_t* timing, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N) return;

    uint64_t start = clock64();
    int sum, count;
    run_udf(input[tid], &sum, &count);
    output[tid] = sum;
    uint64_t end = clock64();

    timing[tid] = end - start;
}

int main() {
    const int N = 64;
    int h_input[N];
    int h_output[N];
    uint64_t h_timing[N];

    srand(time(NULL));
    for (int i = 0; i < N; ++i)
        h_input[i] = rand() % 1000 + 100;  // Random input = divergence

    int *d_input, *d_output;
    uint64_t* d_timing;
    hipMalloc(&d_input, sizeof(int) * N);
    hipMalloc(&d_output, sizeof(int) * N);
    hipMalloc(&d_timing, sizeof(uint64_t) * N);

    hipMemcpy(d_input, h_input, sizeof(int) * N, hipMemcpyHostToDevice);

    profile_udf_kernel<<<1, N>>>(d_input, d_output, d_timing, N);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(h_timing, d_timing, sizeof(uint64_t) * N, hipMemcpyDeviceToHost);

    std::cout << "\n=== Divergent Execution Times ===\n";
    for (int i = 0; i < N; ++i)
        std::cout << "Thread " << i << " | Input: " << h_input[i]
                  << " | Time: " << h_timing[i] << " cycles\n";

    hipFree(d_input); hipFree(d_output); hipFree(d_timing);
    return 0;
}
